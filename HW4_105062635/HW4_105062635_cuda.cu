#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int INF = 10000000;
const int NS = 16;
void input(char *inFileName);
void output(char *outFileName);

int n, m;	// Number of vertices, edges
int *Dist;

bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);
	int i;
	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			hipGetDeviceProperties(&prop, i);
			printf("Device Number: %d\n", i);
			printf("  Device name: %s\n", prop.name);
			printf("  Major: %d Overlap: %d\n", prop.major, prop.deviceOverlap);
			printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
			printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
			printf("  Shared Memory Size per Block (bytes): %d\n", prop.sharedMemPerBlock);
			printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		}
	}
	return true;
}

void input(char *inFileName)
{
	FILE *infile = fopen(inFileName, "r");
	fscanf(infile, "%d %d", &n, &m);
	hipHostMalloc ((void**) &Dist, n*n*sizeof(int));

	for (int i = 0; i < n*n; ++i) {
		if (i/n == i%n)	Dist[i] = 0;
		else Dist[i] = INF;
	}

	while (--m >= 0) {
		int a, b, v;
		fscanf(infile, "%d %d %d", &a, &b, &v);
		--a, --b;
		Dist[a*n + b] = v;
	}
}

void output(char *outFileName)
{
	FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < n*n; ++i) {
		if(Dist[i] >= INF) fprintf(outfile, "INF ");
		else fprintf(outfile, "%d ", Dist[i]);
		if(i%n == n-1)fprintf(outfile, "\n");
	}
}

__host__ __device__ int ceil(int a, int b)
{
	return (a + b -1)/b;
}

__global__ void block_FW1_Kernel(int B, int r, int n, int *d_Dist)
{
	extern __shared__ int b_Dist[];
	int xi, yi, x1, x2, x3, x4, y1, y2, y3, y4;
	bool within1, within2, within3, within4;
	xi = threadIdx.x%B;
	yi = threadIdx.x/B;
	bool within = r*B+yi<n && r*B+xi<n;
	if(B<=32){
		if(within)b_Dist[yi*B+xi] = d_Dist[(r*B+yi)*n+r*B+xi];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n ; ++k) {
			if(within && b_Dist[yi*B+xi] > b_Dist[k*B+xi] + b_Dist[yi*B+k])
				b_Dist[yi*B+xi] = b_Dist[k*B+xi] + b_Dist[yi*B+k];
			__syncthreads();
		}
		if(within)d_Dist[(r*B+yi)*n+r*B+xi] = b_Dist[yi*B+xi];
	}
	else if(B<=64){
		x1 = xi;
		y1 = yi;
		within1 = r*B+y1<n && r*B+x1<n && y1<B;
		y2 = y1+(x1+1024)/B;
		x2 = (x1+1024)%B;
		within2 = r*B+y2<n && r*B+x2<n && y2<B;
		y3 = y2+(x2+1024)/B;
		x3 = (x2+1024)%B;
		within3 = r*B+y3<n && r*B+x3<n && y3<B;
		y4 = y3+(x3+1024)/B;
		x4 = (x3+1024)%B;
		within4 = r*B+y4<n && r*B+x4<n && y4<B;
		if(within1)b_Dist[y1*B+x1] = d_Dist[(r*B+y1)*n+r*B+x1];
		if(within2)b_Dist[y2*B+x2] = d_Dist[(r*B+y2)*n+r*B+x2];
		if(within3)b_Dist[y3*B+x3] = d_Dist[(r*B+y3)*n+r*B+x3];
		if(within4)b_Dist[y4*B+x4] = d_Dist[(r*B+y4)*n+r*B+x4];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n ; ++k) {
			if(within1 && b_Dist[y1*B+x1] > b_Dist[k*B+x1] + b_Dist[y1*B+k])
				b_Dist[y1*B+x1] = b_Dist[k*B+x1] + b_Dist[y1*B+k];
			if(within2 && b_Dist[y2*B+x2] > b_Dist[k*B+x2] + b_Dist[y2*B+k])
				b_Dist[y2*B+x2] = b_Dist[k*B+x2] + b_Dist[y2*B+k];
			if(within3 && b_Dist[y3*B+x3] > b_Dist[k*B+x3] + b_Dist[y3*B+k])
				b_Dist[y3*B+x3] = b_Dist[k*B+x3] + b_Dist[y3*B+k];
			if(within4 && b_Dist[y4*B+x4] > b_Dist[k*B+x4] + b_Dist[y4*B+k])
				b_Dist[y4*B+x4] = b_Dist[k*B+x4] + b_Dist[y4*B+k];
			__syncthreads();
		}
		if(within1)d_Dist[(r*B+y1)*n+r*B+x1] = b_Dist[y1*B+x1];
		if(within2)d_Dist[(r*B+y2)*n+r*B+x2] = b_Dist[y2*B+x2];
		if(within3)d_Dist[(r*B+y3)*n+r*B+x3] = b_Dist[y3*B+x3];
		if(within4)d_Dist[(r*B+y4)*n+r*B+x4] = b_Dist[y4*B+x4];
	}
	else{
		int iter = ceil(B*B, 1024);
		for (int k = 0; k < B && r*B+k<n; ++k) {
			xi = threadIdx.x%B;
			yi = threadIdx.x/B;
			for(int i=0; i<iter; i++){
				within = r*B+yi<n && r*B+xi<n && yi<B;
				if(within && d_Dist[(r*B+yi)*n+r*B+xi] > d_Dist[(r*B+k)*n+r*B+xi] + d_Dist[(r*B+yi)*n+r*B+k])
					d_Dist[(r*B+yi)*n+r*B+xi] = d_Dist[(r*B+k)*n+r*B+xi] + d_Dist[(r*B+yi)*n+r*B+k];
				yi = yi+(xi+1024)/B;
				xi = (xi+1024)%B;
			}
			__syncthreads();
		}
	}
}

__global__ void block_FW2_Kernel(int B, int r, int n, int *d_Dist)
{
	extern __shared__ int b_Dist[];
	int xi, yi, xb, yb, x1, x2, x3, x4, y1, y2, y3, y4;
	bool within1, within2, within3, within4;
	int width = ceil(n, B);
	xb = blockIdx.x < width ? blockIdx.x : r;
	yb = blockIdx.x < width ? r : blockIdx.x - width;
	xi = threadIdx.x%B;
	yi = threadIdx.x/B;
	bool within = yb*B+yi<n && xb*B+xi<n && !(xb==r && yb==r);  
	if(B<=32){
		if(within)b_Dist[yi*B+xi] = d_Dist[(yb*B+yi)*n+xb*B+xi];
		if(r*B+yi<n && r*B+xi<n)b_Dist[B*B+yi*B+xi] = d_Dist[(r*B+yi)*n+r*B+xi];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n; ++k) {
			if(blockIdx.x>=width && within && b_Dist[yi*B+xi] > b_Dist[B*B+k*B+xi] + b_Dist[yi*B+k])
				b_Dist[yi*B+xi] = b_Dist[B*B+k*B+xi] + b_Dist[yi*B+k];
			else if(blockIdx.x<width && within && b_Dist[yi*B+xi] > b_Dist[B*B+yi*B+k] + b_Dist[k*B+xi])
				b_Dist[yi*B+xi] = b_Dist[B*B+yi*B+k] + b_Dist[k*B+xi];
			__syncthreads();
		}
		if(within)d_Dist[(yb*B+yi)*n+xb*B+xi] = b_Dist[yi*B+xi];
	}
	else if(B<=64){
		x1 = xi;
		y1 = yi;
		within1 = yb*B+y1<n && xb*B+x1<n && !(xb==r && yb==r) && y1<B;  
		y2 = y1+(x1+1024)/B;
		x2 = (x1+1024)%B;
		within2 = yb*B+y2<n && xb*B+x2<n && !(xb==r && yb==r) && y2<B;  
		y3 = y2+(x2+1024)/B;
		x3 = (x2+1024)%B;
		within3 = yb*B+y3<n && xb*B+x3<n && !(xb==r && yb==r) && y3<B;  
		y4 = y3+(x3+1024)/B;
		x4 = (x3+1024)%B;
		within4 = yb*B+y4<n && xb*B+x4<n && !(xb==r && yb==r) && y4<B;  
		if(within1)b_Dist[y1*B+x1] = d_Dist[(yb*B+y1)*n+xb*B+x1];
		if(r*B+y1<n && r*B+x1<n && y1<B)b_Dist[B*B+y1*B+x1] = d_Dist[(r*B+y1)*n+r*B+x1];
		if(within2)b_Dist[y2*B+x2] = d_Dist[(yb*B+y2)*n+xb*B+x2];
		if(r*B+y2<n && r*B+x2<n && y2<B)b_Dist[B*B+y2*B+x2] = d_Dist[(r*B+y2)*n+r*B+x2];
		if(within3)b_Dist[y3*B+x3] = d_Dist[(yb*B+y3)*n+xb*B+x3];
		if(r*B+y3<n && r*B+x3<n && y3<B)b_Dist[B*B+y3*B+x3] = d_Dist[(r*B+y3)*n+r*B+x3];
		if(within4)b_Dist[y4*B+x4] = d_Dist[(yb*B+y4)*n+xb*B+x4];
		if(r*B+y4<n && r*B+x4<n && y4<B)b_Dist[B*B+y4*B+x4] = d_Dist[(r*B+y4)*n+r*B+x4];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n; ++k) {
			if(blockIdx.x>=width){
				if(within1 && b_Dist[y1*B+x1] > b_Dist[B*B+k*B+x1] + b_Dist[y1*B+k])
					b_Dist[y1*B+x1] = b_Dist[B*B+k*B+x1] + b_Dist[y1*B+k];
				if(within2 && b_Dist[y2*B+x2] > b_Dist[B*B+k*B+x2] + b_Dist[y2*B+k])
					b_Dist[y2*B+x2] = b_Dist[B*B+k*B+x2] + b_Dist[y2*B+k];
				if(within3 && b_Dist[y3*B+x3] > b_Dist[B*B+k*B+x3] + b_Dist[y3*B+k])
					b_Dist[y3*B+x3] = b_Dist[B*B+k*B+x3] + b_Dist[y3*B+k];
				if(within4 && b_Dist[y4*B+x4] > b_Dist[B*B+k*B+x4] + b_Dist[y4*B+k])
					b_Dist[y4*B+x4] = b_Dist[B*B+k*B+x4] + b_Dist[y4*B+k];
			}
			else {
				if(within1 && b_Dist[y1*B+x1] > b_Dist[B*B+y1*B+k] + b_Dist[k*B+x1])
					b_Dist[y1*B+x1] = b_Dist[B*B+y1*B+k] + b_Dist[k*B+x1];
				if(within2 && b_Dist[y2*B+x2] > b_Dist[B*B+y2*B+k] + b_Dist[k*B+x2])
					b_Dist[y2*B+x2] = b_Dist[B*B+y2*B+k] + b_Dist[k*B+x2];
				if(within3 && b_Dist[y3*B+x3] > b_Dist[B*B+y3*B+k] + b_Dist[k*B+x3])
					b_Dist[y3*B+x3] = b_Dist[B*B+y3*B+k] + b_Dist[k*B+x3];
				if(within4 && b_Dist[y4*B+x4] > b_Dist[B*B+y4*B+k] + b_Dist[k*B+x4])
					b_Dist[y4*B+x4] = b_Dist[B*B+y4*B+k] + b_Dist[k*B+x4];
			}
			__syncthreads();
		}
		if(within1)d_Dist[(yb*B+y1)*n+xb*B+x1] = b_Dist[y1*B+x1];
		if(within2)d_Dist[(yb*B+y2)*n+xb*B+x2] = b_Dist[y2*B+x2];
		if(within3)d_Dist[(yb*B+y3)*n+xb*B+x3] = b_Dist[y3*B+x3];
		if(within4)d_Dist[(yb*B+y4)*n+xb*B+x4] = b_Dist[y4*B+x4];
	}
	else{
		int iter = ceil(B*B, 1024);
		for (int k = 0; k < B && r*B+k<n; ++k) {
			xi = threadIdx.x%B;
			yi = threadIdx.x/B;
			for(int i=0; i<iter; i++){
				within = yb*B+yi<n && xb*B+xi<n && !(xb==r && yb==r) && yi<B;  
				if(blockIdx.x>=width && within && d_Dist[(yb*B+yi)*n+xb*B+xi] > d_Dist[(r*B+k)*n+r*B+xi] + d_Dist[(yb*B+yi)*n+xb*B+k])
					d_Dist[(yb*B+yi)*n+xb*B+xi] = d_Dist[(r*B+k)*n+r*B+xi] + d_Dist[(yb*B+yi)*n+xb*B+k];
				else if(blockIdx.x<width && within && d_Dist[(yb*B+yi)*n+xb*B+xi] > d_Dist[(r*B+yi)*n+r*B+k] + d_Dist[(yb*B+k)*n+xb*B+xi])
					d_Dist[(yb*B+yi)*n+xb*B+xi] = d_Dist[(r*B+yi)*n+r*B+k] + d_Dist[(yb*B+k)*n+xb*B+xi];
				yi = yi+(xi+1024)/B;
				xi = (xi+1024)%B;
			}
			__syncthreads();
		}
	}
}

__global__ void block_FW3_Kernel(int B, int r, int n, int boff, int *d_Dist)
{
	extern __shared__ int b_Dist[];
	int xi, yi, xb, yb, x1, x2, x3, x4, y1, y2, y3, y4;
	bool within1, within2, within3, within4;
	xb = blockIdx.x;
	yb = blockIdx.y + boff;
	xi = threadIdx.x%B;
	yi = threadIdx.x/B;
	bool within = yb*B+yi<n && xb*B+xi<n && xb!=r && yb !=r; 
	if(B<=32){
		if(within)b_Dist[yi*B+xi] = d_Dist[(yb*B+yi)*n+xb*B+xi];
		if(r*B+yi<n && xb*B+xi<n)b_Dist[B*B+yi*B+xi] = d_Dist[(r*B+yi)*n+xb*B+xi];
		if(yb*B+yi<n && r*B+xi<n)b_Dist[2*B*B+yi*B+xi] = d_Dist[(yb*B+yi)*n+r*B+xi];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n; ++k) {
			if(within && b_Dist[yi*B+xi] > b_Dist[B*B+k*B+xi] + b_Dist[2*B*B+yi*B+k])
				b_Dist[yi*B+xi] = b_Dist[B*B+k*B+xi] + b_Dist[2*B*B+yi*B+k];
			__syncthreads();
		}
		if(within)d_Dist[(yb*B+yi)*n+xb*B+xi] = b_Dist[yi*B+xi];
	}
	else if(B<=64){
		x1 = xi;
		y1 = yi;
		within1 = yb*B+y1<n && xb*B+x1<n && xb!=r && yb!=r && y1<B; 
		y2 = y1+(x1+1024)/B;
		x2 = (x1+1024)%B;
		within2 = yb*B+y2<n && xb*B+x2<n && xb!=r && yb!=r && y2<B; 
		y3 = y2+(x2+1024)/B;
		x3 = (x2+1024)%B;
		within3 = yb*B+y3<n && xb*B+x3<n && xb!=r && yb!=r && y3<B; 
		y4 = y3+(x3+1024)/B;
		x4 = (x3+1024)%B;
		within4 = yb*B+y4<n && xb*B+x4<n && xb!=r && yb!=r && y4<B; 
		if(within1)b_Dist[y1*B+x1] = d_Dist[(yb*B+y1)*n+xb*B+x1];
		if(r*B+y1<n && xb*B+x1<n && y1<B)b_Dist[B*B+y1*B+x1] = d_Dist[(r*B+y1)*n+xb*B+x1];
		if(yb*B+y1<n && r*B+x1<n && y1<B)b_Dist[2*B*B+y1*B+x1] = d_Dist[(yb*B+y1)*n+r*B+x1];
		if(within2)b_Dist[y2*B+x2] = d_Dist[(yb*B+y2)*n+xb*B+x2];
		if(r*B+y2<n && xb*B+x2<n && y2<B)b_Dist[B*B+y2*B+x2] = d_Dist[(r*B+y2)*n+xb*B+x2];
		if(yb*B+y2<n && r*B+x2<n && y2<B)b_Dist[2*B*B+y2*B+x2] = d_Dist[(yb*B+y2)*n+r*B+x2];
		if(within3)b_Dist[y3*B+x3] = d_Dist[(yb*B+y3)*n+xb*B+x3];
		if(r*B+y3<n && xb*B+x3<n && y3<B)b_Dist[B*B+y3*B+x3] = d_Dist[(r*B+y3)*n+xb*B+x3];
		if(yb*B+y3<n && r*B+x3<n && y3<B)b_Dist[2*B*B+y3*B+x3] = d_Dist[(yb*B+y3)*n+r*B+x3];
		if(within4)b_Dist[y4*B+x4] = d_Dist[(yb*B+y4)*n+xb*B+x4];
		if(r*B+y4<n && xb*B+x4<n && y4<B)b_Dist[B*B+y4*B+x4] = d_Dist[(r*B+y4)*n+xb*B+x4];
		if(yb*B+y4<n && r*B+x4<n && y4<B)b_Dist[2*B*B+y4*B+x4] = d_Dist[(yb*B+y4)*n+r*B+x4];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n; ++k) {
			if(within1 && b_Dist[y1*B+x1] > b_Dist[B*B+k*B+x1] + b_Dist[2*B*B+y1*B+k])
				b_Dist[y1*B+x1] = b_Dist[B*B+k*B+x1] + b_Dist[2*B*B+y1*B+k];
			if(within2 && b_Dist[y2*B+x2] > b_Dist[B*B+k*B+x2] + b_Dist[2*B*B+y2*B+k])
				b_Dist[y2*B+x2] = b_Dist[B*B+k*B+x2] + b_Dist[2*B*B+y2*B+k];
			if(within3 && b_Dist[y3*B+x3] > b_Dist[B*B+k*B+x3] + b_Dist[2*B*B+y3*B+k])
				b_Dist[y3*B+x3] = b_Dist[B*B+k*B+x3] + b_Dist[2*B*B+y3*B+k];
			if(within4 && b_Dist[y4*B+x4] > b_Dist[B*B+k*B+x4] + b_Dist[2*B*B+y4*B+k])
				b_Dist[y4*B+x4] = b_Dist[B*B+k*B+x4] + b_Dist[2*B*B+y4*B+k];
			__syncthreads();
		}
		if(within1)d_Dist[(yb*B+y1)*n+xb*B+x1] = b_Dist[y1*B+x1];
		if(within2)d_Dist[(yb*B+y2)*n+xb*B+x2] = b_Dist[y2*B+x2];
		if(within3)d_Dist[(yb*B+y3)*n+xb*B+x3] = b_Dist[y3*B+x3];
		if(within4)d_Dist[(yb*B+y4)*n+xb*B+x4] = b_Dist[y4*B+x4];
	}
	else{
		int iter = ceil(B*B, 1024);
		for (int k = 0; k < B && r*B+k<n; ++k) {
			xi = threadIdx.x%B;
			yi = threadIdx.x/B;
			for(int i=0; i<iter; i++){
				within = yb*B+yi<n && xb*B+xi<n && xb!=r && yb !=r && yi<B; 
				if(within && d_Dist[(yb*B+yi)*n+xb*B+xi] > d_Dist[(r*B+k)*n+xb*B+xi] + d_Dist[(yb*B+yi)*n+r*B+k])
					d_Dist[(yb*B+yi)*n+xb*B+xi] = d_Dist[(r*B+k)*n+xb*B+xi] + d_Dist[(yb*B+yi)*n+r*B+k];
					yi = yi+(xi+1024)/B;
					xi = (xi+1024)%B;
			}
			 __syncthreads();
		}
	}
}

int main(int argc, char* argv[])
{
	//InitCUDA();
	//hipEvent_t start, stop;
	//float time;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	input(argv[1]);
	int B = atoi(argv[3]) > n ? n : atoi(argv[3]);
	int NT = B > 32 ? 32*32 : B*B;
	int NB = B > 64 ? 64*64 : B*B;
	int *d_Dist;
	hipStream_t streams[NS];
	for(int i=0; i<NS; i++)hipStreamCreate(&streams[i]);
	hipMalloc(&d_Dist, n*n*sizeof(int));
	hipMemcpy(d_Dist, Dist, n*n*sizeof(int), hipMemcpyHostToDevice);
	int round = ceil(n, B);
	int bps = round == 1 ? 1 : ceil(round-1, NS-1); //# blocks per stream
	int NSused = ceil(round-1, bps); //# streams really needed
	//printf("round:%d bps:%d used:%d\n",round,bps,NSused);
	dim3 block(round, round);
	dim3 block2(round, bps); 
	//hipEventRecord(start);
	for (int r = 0; r < round; ++r) {	
		block_FW1_Kernel<<<1, NT, NB*sizeof(int)>>>(B, r, n, d_Dist);
		block_FW2_Kernel<<<2*round, NT, 2*NB*sizeof(int)>>>(B, r, n, d_Dist);
		if(r == round-1){
			hipMemcpyAsync(Dist+r*B*n, d_Dist+r*B*n, (n-r*B)*n*sizeof(int), hipMemcpyDeviceToHost, streams[NSused]); 
			for(int i=0; i<NSused-1; i++){
				block_FW3_Kernel<<<block2, NT, 3*NB*sizeof(int), streams[i]>>>(B, r, n, i*bps, d_Dist);
				hipMemcpyAsync(Dist+n*bps*B*i, d_Dist+n*bps*B*i, n*bps*B*sizeof(int), hipMemcpyDeviceToHost, streams[i]); 
			}
			block_FW3_Kernel<<<block2, NT, 3*NB*sizeof(int), streams[NSused-1]>>>(B, r, n, (NSused-1)*bps, d_Dist);
			int remain = (round-1)%bps == 0 ? bps : (round-1)%bps;
			hipMemcpyAsync(Dist+n*bps*B*(NSused-1), d_Dist+n*bps*B*(NSused-1), n*(remain*B)*sizeof(int), hipMemcpyDeviceToHost, streams[NSused-1]); 
		}
		else block_FW3_Kernel<<<block, NT, 3*NB*sizeof(int)>>>(B, r, n, 0, d_Dist);
	}	
	for(int i=0; i<NS; i++)hipStreamSynchronize(streams[i]);
	//hipMemcpy(Dist, d_Dist, n*n*sizeof(int), hipMemcpyDeviceToHost); 
	//hipEventRecord(stop);
	//hipEventSynchronize(stop);
	hipDeviceSynchronize();
	output(argv[2]);
	for(int i=0; i<NS; i++)hipStreamDestroy(streams[i]);
	//hipEventElapsedTime(&time, start, stop);
	//hipEventDestroy(start);
	//hipEventDestroy(stop);
	hipFree(d_Dist);
	hipHostFree(Dist);
	//printf("time elapsed for calculation: %lf!!!\n", time);
	return 0;
}

