#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

const int INF = 10000000;
const int NS = 10;
void input(char *inFileName);
void output(char *outFileName);

int n, m;	// Number of vertices, edges
int *Dist;

bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);
	int i;
	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			hipGetDeviceProperties(&prop, i);
			printf("Device Number: %d\n", i);
			printf("  Device name: %s\n", prop.name);
			printf("  Major: %d Overlap: %d\n", prop.major, prop.deviceOverlap);
			printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
			printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
			printf("  Shared Memory Size per Block (bytes): %d\n", prop.sharedMemPerBlock);
			printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		}
	}
	return true;
}

void input(char *inFileName)
{
	FILE *infile = fopen(inFileName, "r");
	fscanf(infile, "%d %d", &n, &m);
	hipHostMalloc ((void**) &Dist, n*n*sizeof(int));

	for (int i = 0; i < n*n; ++i) {
		if (i/n == i%n)	Dist[i] = 0;
		else Dist[i] = INF;
	}

	while (--m >= 0) {
		int a, b, v;
		fscanf(infile, "%d %d %d", &a, &b, &v);
		--a, --b;
		Dist[a*n + b] = v;
	}
}

void output(char *outFileName)
{
	FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < n*n; ++i) {
		if(Dist[i] >= INF) fprintf(outfile, "INF ");
		else fprintf(outfile, "%d ", Dist[i]);
		if(i%n == n-1)fprintf(outfile, "\n");
	}
}

__host__ __device__ int ceil(int a, int b)
{
	return (a + b -1)/b;
}

__global__ void block_FW1_Kernel(int B, int r, int n, int *d_Dist)
{
	extern __shared__ int b_Dist[];
	int xi, yi, x1, x2, x3, x4, y1, y2, y3, y4;
	bool within1, within2, within3, within4;
	xi = threadIdx.x%B;
	yi = threadIdx.x/B;
	bool within = r*B+yi<n && r*B+xi<n;
	if(B<=32){
		if(within)b_Dist[yi*B+xi] = d_Dist[(r*B+yi)*n+r*B+xi];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n ; ++k) {
			if(within && b_Dist[yi*B+xi] > b_Dist[k*B+xi] + b_Dist[yi*B+k])
				b_Dist[yi*B+xi] = b_Dist[k*B+xi] + b_Dist[yi*B+k];
			__syncthreads();
		}
		if(within)d_Dist[(r*B+yi)*n+r*B+xi] = b_Dist[yi*B+xi];
	}
	else if(B<=64){
		x1 = xi;
		y1 = yi;
		within1 = r*B+y1<n && r*B+x1<n && y1<B;
		y2 = y1+(x1+1024)/B;
		x2 = (x1+1024)%B;
		within2 = r*B+y2<n && r*B+x2<n && y2<B;
		y3 = y2+(x2+1024)/B;
		x3 = (x2+1024)%B;
		within3 = r*B+y3<n && r*B+x3<n && y3<B;
		y4 = y3+(x3+1024)/B;
		x4 = (x3+1024)%B;
		within4 = r*B+y4<n && r*B+x4<n && y4<B;
		if(within1)b_Dist[y1*B+x1] = d_Dist[(r*B+y1)*n+r*B+x1];
		if(within2)b_Dist[y2*B+x2] = d_Dist[(r*B+y2)*n+r*B+x2];
		if(within3)b_Dist[y3*B+x3] = d_Dist[(r*B+y3)*n+r*B+x3];
		if(within4)b_Dist[y4*B+x4] = d_Dist[(r*B+y4)*n+r*B+x4];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n ; ++k) {
			if(within1 && b_Dist[y1*B+x1] > b_Dist[k*B+x1] + b_Dist[y1*B+k])
				b_Dist[y1*B+x1] = b_Dist[k*B+x1] + b_Dist[y1*B+k];
			if(within2 && b_Dist[y2*B+x2] > b_Dist[k*B+x2] + b_Dist[y2*B+k])
				b_Dist[y2*B+x2] = b_Dist[k*B+x2] + b_Dist[y2*B+k];
			if(within3 && b_Dist[y3*B+x3] > b_Dist[k*B+x3] + b_Dist[y3*B+k])
				b_Dist[y3*B+x3] = b_Dist[k*B+x3] + b_Dist[y3*B+k];
			if(within4 && b_Dist[y4*B+x4] > b_Dist[k*B+x4] + b_Dist[y4*B+k])
				b_Dist[y4*B+x4] = b_Dist[k*B+x4] + b_Dist[y4*B+k];
			__syncthreads();
		}
		if(within1)d_Dist[(r*B+y1)*n+r*B+x1] = b_Dist[y1*B+x1];
		if(within2)d_Dist[(r*B+y2)*n+r*B+x2] = b_Dist[y2*B+x2];
		if(within3)d_Dist[(r*B+y3)*n+r*B+x3] = b_Dist[y3*B+x3];
		if(within4)d_Dist[(r*B+y4)*n+r*B+x4] = b_Dist[y4*B+x4];
	}
	else{
		int iter = ceil(B*B, 1024);
		for (int k = 0; k < B && r*B+k<n; ++k) {
			xi = threadIdx.x%B;
			yi = threadIdx.x/B;
			for(int i=0; i<iter; i++){
				within = r*B+yi<n && r*B+xi<n && yi<B;
				if(within && d_Dist[(r*B+yi)*n+r*B+xi] > d_Dist[(r*B+k)*n+r*B+xi] + d_Dist[(r*B+yi)*n+r*B+k])
					d_Dist[(r*B+yi)*n+r*B+xi] = d_Dist[(r*B+k)*n+r*B+xi] + d_Dist[(r*B+yi)*n+r*B+k];
				yi = yi+(xi+1024)/B;
				xi = (xi+1024)%B;
			}
			__syncthreads();
		}
	}
}

__global__ void block_FW2_Kernel(int B, int r, int n, int *d_Dist)
{
	extern __shared__ int b_Dist[];
	int xi, yi, xb, yb, x1, x2, x3, x4, y1, y2, y3, y4;
	bool within1, within2, within3, within4;
	int width = ceil(n, B);
	xb = blockIdx.x < width ? blockIdx.x : r;
	yb = blockIdx.x < width ? r : blockIdx.x - width;
	xi = threadIdx.x%B;
	yi = threadIdx.x/B;
	bool within = yb*B+yi<n && xb*B+xi<n && !(xb==r && yb==r);  
	if(B<=32){
		if(within)b_Dist[yi*B+xi] = d_Dist[(yb*B+yi)*n+xb*B+xi];
		if(r*B+yi<n && r*B+xi<n)b_Dist[B*B+yi*B+xi] = d_Dist[(r*B+yi)*n+r*B+xi];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n; ++k) {
			if(blockIdx.x>=width && within && b_Dist[yi*B+xi] > b_Dist[B*B+k*B+xi] + b_Dist[yi*B+k])
				b_Dist[yi*B+xi] = b_Dist[B*B+k*B+xi] + b_Dist[yi*B+k];
			else if(blockIdx.x<width && within && b_Dist[yi*B+xi] > b_Dist[B*B+yi*B+k] + b_Dist[k*B+xi])
				b_Dist[yi*B+xi] = b_Dist[B*B+yi*B+k] + b_Dist[k*B+xi];
			__syncthreads();
		}
		if(within)d_Dist[(yb*B+yi)*n+xb*B+xi] = b_Dist[yi*B+xi];
	}
	else if(B<=64){
		x1 = xi;
		y1 = yi;
		within1 = yb*B+y1<n && xb*B+x1<n && !(xb==r && yb==r) && y1<B;  
		y2 = y1+(x1+1024)/B;
		x2 = (x1+1024)%B;
		within2 = yb*B+y2<n && xb*B+x2<n && !(xb==r && yb==r) && y2<B;  
		y3 = y2+(x2+1024)/B;
		x3 = (x2+1024)%B;
		within3 = yb*B+y3<n && xb*B+x3<n && !(xb==r && yb==r) && y3<B;  
		y4 = y3+(x3+1024)/B;
		x4 = (x3+1024)%B;
		within4 = yb*B+y4<n && xb*B+x4<n && !(xb==r && yb==r) && y4<B;  
		if(within1)b_Dist[y1*B+x1] = d_Dist[(yb*B+y1)*n+xb*B+x1];
		if(r*B+y1<n && r*B+x1<n && y1<B)b_Dist[B*B+y1*B+x1] = d_Dist[(r*B+y1)*n+r*B+x1];
		if(within2)b_Dist[y2*B+x2] = d_Dist[(yb*B+y2)*n+xb*B+x2];
		if(r*B+y2<n && r*B+x2<n && y2<B)b_Dist[B*B+y2*B+x2] = d_Dist[(r*B+y2)*n+r*B+x2];
		if(within3)b_Dist[y3*B+x3] = d_Dist[(yb*B+y3)*n+xb*B+x3];
		if(r*B+y3<n && r*B+x3<n && y3<B)b_Dist[B*B+y3*B+x3] = d_Dist[(r*B+y3)*n+r*B+x3];
		if(within4)b_Dist[y4*B+x4] = d_Dist[(yb*B+y4)*n+xb*B+x4];
		if(r*B+y4<n && r*B+x4<n && y4<B)b_Dist[B*B+y4*B+x4] = d_Dist[(r*B+y4)*n+r*B+x4];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n; ++k) {
			if(blockIdx.x>=width){
				if(within1 && b_Dist[y1*B+x1] > b_Dist[B*B+k*B+x1] + b_Dist[y1*B+k])
					b_Dist[y1*B+x1] = b_Dist[B*B+k*B+x1] + b_Dist[y1*B+k];
				if(within2 && b_Dist[y2*B+x2] > b_Dist[B*B+k*B+x2] + b_Dist[y2*B+k])
					b_Dist[y2*B+x2] = b_Dist[B*B+k*B+x2] + b_Dist[y2*B+k];
				if(within3 && b_Dist[y3*B+x3] > b_Dist[B*B+k*B+x3] + b_Dist[y3*B+k])
					b_Dist[y3*B+x3] = b_Dist[B*B+k*B+x3] + b_Dist[y3*B+k];
				if(within4 && b_Dist[y4*B+x4] > b_Dist[B*B+k*B+x4] + b_Dist[y4*B+k])
					b_Dist[y4*B+x4] = b_Dist[B*B+k*B+x4] + b_Dist[y4*B+k];
			}
			else {
				if(within1 && b_Dist[y1*B+x1] > b_Dist[B*B+y1*B+k] + b_Dist[k*B+x1])
					b_Dist[y1*B+x1] = b_Dist[B*B+y1*B+k] + b_Dist[k*B+x1];
				if(within2 && b_Dist[y2*B+x2] > b_Dist[B*B+y2*B+k] + b_Dist[k*B+x2])
					b_Dist[y2*B+x2] = b_Dist[B*B+y2*B+k] + b_Dist[k*B+x2];
				if(within3 && b_Dist[y3*B+x3] > b_Dist[B*B+y3*B+k] + b_Dist[k*B+x3])
					b_Dist[y3*B+x3] = b_Dist[B*B+y3*B+k] + b_Dist[k*B+x3];
				if(within4 && b_Dist[y4*B+x4] > b_Dist[B*B+y4*B+k] + b_Dist[k*B+x4])
					b_Dist[y4*B+x4] = b_Dist[B*B+y4*B+k] + b_Dist[k*B+x4];
			}
			__syncthreads();
		}
		if(within1)d_Dist[(yb*B+y1)*n+xb*B+x1] = b_Dist[y1*B+x1];
		if(within2)d_Dist[(yb*B+y2)*n+xb*B+x2] = b_Dist[y2*B+x2];
		if(within3)d_Dist[(yb*B+y3)*n+xb*B+x3] = b_Dist[y3*B+x3];
		if(within4)d_Dist[(yb*B+y4)*n+xb*B+x4] = b_Dist[y4*B+x4];
	}
	else{
		int iter = ceil(B*B, 1024);
		for (int k = 0; k < B && r*B+k<n; ++k) {
			xi = threadIdx.x%B;
			yi = threadIdx.x/B;
			for(int i=0; i<iter; i++){
				within = yb*B+yi<n && xb*B+xi<n && !(xb==r && yb==r) && yi<B;  
				if(blockIdx.x>=width && within && d_Dist[(yb*B+yi)*n+xb*B+xi] > d_Dist[(r*B+k)*n+r*B+xi] + d_Dist[(yb*B+yi)*n+xb*B+k])
					d_Dist[(yb*B+yi)*n+xb*B+xi] = d_Dist[(r*B+k)*n+r*B+xi] + d_Dist[(yb*B+yi)*n+xb*B+k];
				else if(blockIdx.x<width && within && d_Dist[(yb*B+yi)*n+xb*B+xi] > d_Dist[(r*B+yi)*n+r*B+k] + d_Dist[(yb*B+k)*n+xb*B+xi])
					d_Dist[(yb*B+yi)*n+xb*B+xi] = d_Dist[(r*B+yi)*n+r*B+k] + d_Dist[(yb*B+k)*n+xb*B+xi];
				yi = yi+(xi+1024)/B;
				xi = (xi+1024)%B;
			}
			__syncthreads();
		}
	}
}

__global__ void block_FW3_Kernel(int B, int r, int n, int boff, int *d_Dist)
{
	extern __shared__ int b_Dist[];
	int xi, yi, xb, yb, x1, x2, x3, x4, y1, y2, y3, y4;
	bool within1, within2, within3, within4;
	xb = blockIdx.x;
	yb = blockIdx.y + boff;
	xi = threadIdx.x%B;
	yi = threadIdx.x/B;
	bool within = yb*B+yi<n && xb*B+xi<n && xb!=r && yb !=r; 
	if(B<=32){
		if(within)b_Dist[yi*B+xi] = d_Dist[(yb*B+yi)*n+xb*B+xi];
		if(r*B+yi<n && xb*B+xi<n)b_Dist[B*B+yi*B+xi] = d_Dist[(r*B+yi)*n+xb*B+xi];
		if(yb*B+yi<n && r*B+xi<n)b_Dist[2*B*B+yi*B+xi] = d_Dist[(yb*B+yi)*n+r*B+xi];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n; ++k) {
			if(within && b_Dist[yi*B+xi] > b_Dist[B*B+k*B+xi] + b_Dist[2*B*B+yi*B+k])
				b_Dist[yi*B+xi] = b_Dist[B*B+k*B+xi] + b_Dist[2*B*B+yi*B+k];
			__syncthreads();
		}
		if(within)d_Dist[(yb*B+yi)*n+xb*B+xi] = b_Dist[yi*B+xi];
	}
	else if(B<=64){
		x1 = xi;
		y1 = yi;
		within1 = yb*B+y1<n && xb*B+x1<n && xb!=r && yb!=r && y1<B; 
		y2 = y1+(x1+1024)/B;
		x2 = (x1+1024)%B;
		within2 = yb*B+y2<n && xb*B+x2<n && xb!=r && yb!=r && y2<B; 
		y3 = y2+(x2+1024)/B;
		x3 = (x2+1024)%B;
		within3 = yb*B+y3<n && xb*B+x3<n && xb!=r && yb!=r && y3<B; 
		y4 = y3+(x3+1024)/B;
		x4 = (x3+1024)%B;
		within4 = yb*B+y4<n && xb*B+x4<n && xb!=r && yb!=r && y4<B; 
		if(within1)b_Dist[y1*B+x1] = d_Dist[(yb*B+y1)*n+xb*B+x1];
		if(r*B+y1<n && xb*B+x1<n && y1<B)b_Dist[B*B+y1*B+x1] = d_Dist[(r*B+y1)*n+xb*B+x1];
		if(yb*B+y1<n && r*B+x1<n && y1<B)b_Dist[2*B*B+y1*B+x1] = d_Dist[(yb*B+y1)*n+r*B+x1];
		if(within2)b_Dist[y2*B+x2] = d_Dist[(yb*B+y2)*n+xb*B+x2];
		if(r*B+y2<n && xb*B+x2<n && y2<B)b_Dist[B*B+y2*B+x2] = d_Dist[(r*B+y2)*n+xb*B+x2];
		if(yb*B+y2<n && r*B+x2<n && y2<B)b_Dist[2*B*B+y2*B+x2] = d_Dist[(yb*B+y2)*n+r*B+x2];
		if(within3)b_Dist[y3*B+x3] = d_Dist[(yb*B+y3)*n+xb*B+x3];
		if(r*B+y3<n && xb*B+x3<n && y3<B)b_Dist[B*B+y3*B+x3] = d_Dist[(r*B+y3)*n+xb*B+x3];
		if(yb*B+y3<n && r*B+x3<n && y3<B)b_Dist[2*B*B+y3*B+x3] = d_Dist[(yb*B+y3)*n+r*B+x3];
		if(within4)b_Dist[y4*B+x4] = d_Dist[(yb*B+y4)*n+xb*B+x4];
		if(r*B+y4<n && xb*B+x4<n && y4<B)b_Dist[B*B+y4*B+x4] = d_Dist[(r*B+y4)*n+xb*B+x4];
		if(yb*B+y4<n && r*B+x4<n && y4<B)b_Dist[2*B*B+y4*B+x4] = d_Dist[(yb*B+y4)*n+r*B+x4];
		__syncthreads();
		for (int k = 0; k < B && r*B+k<n; ++k) {
			if(within1 && b_Dist[y1*B+x1] > b_Dist[B*B+k*B+x1] + b_Dist[2*B*B+y1*B+k])
				b_Dist[y1*B+x1] = b_Dist[B*B+k*B+x1] + b_Dist[2*B*B+y1*B+k];
			if(within2 && b_Dist[y2*B+x2] > b_Dist[B*B+k*B+x2] + b_Dist[2*B*B+y2*B+k])
				b_Dist[y2*B+x2] = b_Dist[B*B+k*B+x2] + b_Dist[2*B*B+y2*B+k];
			if(within3 && b_Dist[y3*B+x3] > b_Dist[B*B+k*B+x3] + b_Dist[2*B*B+y3*B+k])
				b_Dist[y3*B+x3] = b_Dist[B*B+k*B+x3] + b_Dist[2*B*B+y3*B+k];
			if(within4 && b_Dist[y4*B+x4] > b_Dist[B*B+k*B+x4] + b_Dist[2*B*B+y4*B+k])
				b_Dist[y4*B+x4] = b_Dist[B*B+k*B+x4] + b_Dist[2*B*B+y4*B+k];
			__syncthreads();
		}
		if(within1)d_Dist[(yb*B+y1)*n+xb*B+x1] = b_Dist[y1*B+x1];
		if(within2)d_Dist[(yb*B+y2)*n+xb*B+x2] = b_Dist[y2*B+x2];
		if(within3)d_Dist[(yb*B+y3)*n+xb*B+x3] = b_Dist[y3*B+x3];
		if(within4)d_Dist[(yb*B+y4)*n+xb*B+x4] = b_Dist[y4*B+x4];
	}
	else{
		int iter = ceil(B*B, 1024);
		for (int k = 0; k < B && r*B+k<n; ++k) {
			xi = threadIdx.x%B;
			yi = threadIdx.x/B;
			for(int i=0; i<iter; i++){
				within = yb*B+yi<n && xb*B+xi<n && xb!=r && yb !=r && yi<B; 
				if(within && d_Dist[(yb*B+yi)*n+xb*B+xi] > d_Dist[(r*B+k)*n+xb*B+xi] + d_Dist[(yb*B+yi)*n+r*B+k])
					d_Dist[(yb*B+yi)*n+xb*B+xi] = d_Dist[(r*B+k)*n+xb*B+xi] + d_Dist[(yb*B+yi)*n+r*B+k];
				yi = yi+(xi+1024)/B;
				xi = (xi+1024)%B;
			}
			__syncthreads();
		}
	}
}

int main(int argc, char* argv[])
{
	//InitCUDA();
	int task_n, rank;
	bool done[NS];
	MPI_Status status[NS];
	MPI_Request r_req[NS];
	MPI_Request s_req[NS];
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &task_n);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	//double t, t_wait, t_total;
	//t = MPI_Wtime();
	input(argv[1]);
	//printf("input time: %f\n", MPI_Wtime()-t);
	int B = atoi(argv[3]) > n ? n : atoi(argv[3]);
	int NT = B > 32 ? 32*32 : B*B;
	int NB = B > 64 ? 64*64 : B*B;
	int *d_Dist;

	hipSetDevice(rank);
	hipStream_t streams[NS];
	for(int i=0; i<NS; i++)hipStreamCreate(&streams[i]);
	hipMalloc(&d_Dist, n*n*sizeof(int));
	//t = MPI_Wtime();
	hipMemcpy(d_Dist, Dist, n*n*sizeof(int), hipMemcpyHostToDevice);
	//printf("H2D time: %f\n", MPI_Wtime()-t);

	int round = ceil(n, B);
	int round2 = round/2;
	int round1 = round-round2;
	int bps = round1 == 1 ? 1 : ceil(round1, NS-1); //# blocks per stream
	int NSused = ceil(round1, bps); //# streams really needed
	int bps2 = round2 < 1 ? 1 : ceil(round2, NS-1); //# blocks per stream
	int NSused2 = ceil(round2, bps2); //# streams really needed
	//if(rank==0)printf("round:%d bps:%d used:%d\n",round1,bps,NSused);
	//else printf("round2:%d bps2:%d used2:%d\n",round2,bps2,NSused2);
	dim3 block(round, round);
	dim3 block2(round, bps); 
	dim3 block5(round, bps2); 
	dim3 block3(round, round1);
	dim3 block4(round, round2);
	int index, count;
	bool finish = false;
	//t_total = MPI_Wtime();
	for (int r = 0; r < round; ++r) {	
		for(int i=0; i<NS; i++)done[i] = false;
		count = 0;
		finish = false;
		if(rank == 0){
			block_FW1_Kernel<<<1, NT, NB*sizeof(int)>>>(B, r, n, d_Dist);
			block_FW2_Kernel<<<2*round, NT, 2*NB*sizeof(int)>>>(B, r, n, d_Dist);
			if(round2>0){
				for(int i=0; i<NSused2-1; i++){
					MPI_Irecv(Dist+n*(round1*B+bps2*B*i), n*bps2*B, MPI_INT, 1, i, MPI_COMM_WORLD, &r_req[i]);
				}
				MPI_Irecv(Dist+n*(round1*B+bps2*B*(NSused2-1)), n*(n-round1*B-(NSused2-1)*bps2*B), MPI_INT, 1, NSused2-1, MPI_COMM_WORLD, &r_req[NSused2-1]);
			}
			for(int i=0; i<NSused; i++){
				block_FW3_Kernel<<<block2, NT, 3*NB*sizeof(int), streams[i]>>>(B, r, n, i*bps, d_Dist);
				hipMemcpyAsync(Dist+n*bps*B*i, d_Dist+n*bps*B*i, n*bps*B*sizeof(int), hipMemcpyDeviceToHost, streams[i]); 
			}
			if(round2>0){
				while(!finish){
					finish = true;
					for(int i=0; i<NSused; i++){
						if(hipStreamQuery(streams[i])==hipSuccess && !done[i]){
							done[i] = true;
							MPI_Isend(Dist+n*bps*B*i, n*bps*B, MPI_INT, 1, i, MPI_COMM_WORLD, &s_req[i]);
							if(count<NSused2){
								//t = MPI_Wtime();
								MPI_Waitany(NSused2, r_req, &index, status);
								//t_wait += MPI_Wtime()-t;
								if(index == NSused2-1)hipMemcpyAsync(d_Dist+n*(round1*B+bps2*B*(NSused2-1)), Dist+n*(round1*B+bps2*B*(NSused2-1)), n*(n-round1*B-(NSused2-1)*bps2*B)*sizeof(int), hipMemcpyHostToDevice, streams[i]); 
								else hipMemcpyAsync(d_Dist+n*(round1*B+bps2*B*index), Dist+n*(round1*B+bps2*B*index), n*bps2*B*sizeof(int), hipMemcpyHostToDevice, streams[i]); 
							}
							count++;
						}
						if(!done[i])finish = false;
					}
				}
				//t = MPI_Wtime();
				for(int i=0; i<NSused; i++)MPI_Wait(&s_req[i], &status[i]);
				//t_wait += MPI_Wtime()-t;
			}
			for(int i=0; i<NS; i++)hipStreamSynchronize(streams[i]);
		}
		else if(rank == 1 && round2>0){
			block_FW1_Kernel<<<1, NT, NB*sizeof(int)>>>(B, r, n, d_Dist);
			block_FW2_Kernel<<<2*round, NT, 2*NB*sizeof(int)>>>(B, r, n, d_Dist);
			for(int i=0; i<NSused; i++){
				MPI_Irecv(Dist+n*bps*B*i, n*bps*B, MPI_INT, 0, i, MPI_COMM_WORLD, &r_req[i]);
			}
			for(int i=0; i<NSused2-1; i++){
				block_FW3_Kernel<<<block5, NT, 3*NB*sizeof(int), streams[i]>>>(B, r, n, round1+i*bps2, d_Dist);
				hipMemcpyAsync(Dist+n*(round1*B+bps2*B*i), d_Dist+n*(round1*B+bps2*B*i), n*bps2*B*sizeof(int), hipMemcpyDeviceToHost, streams[i]); 
			}
			block_FW3_Kernel<<<block5, NT, 3*NB*sizeof(int), streams[NSused2-1]>>>(B, r, n, round1+(NSused2-1)*bps2, d_Dist);
			hipMemcpyAsync(Dist+n*(round1*B+bps2*B*(NSused2-1)), d_Dist+n*(round1*B+bps2*B*(NSused2-1)), n*(n-round1*B-(NSused2-1)*bps2*B)*sizeof(int), hipMemcpyDeviceToHost, streams[NSused2-1]); 
			while(!finish){
				finish = true;
				for(int i=0; i<NSused2; i++){
					if(hipStreamQuery(streams[i])==hipSuccess && !done[i]){
						done[i]=true;
						if(i == NSused2-1)MPI_Isend(Dist+n*(round1*B+bps2*B*(NSused2-1)), n*(n-round1*B-(NSused2-1)*bps2*B), MPI_INT, 0, NSused2-1, MPI_COMM_WORLD, &s_req[NSused2-1]);
						else MPI_Isend(Dist+n*(round1*B+bps2*B*i), n*bps2*B, MPI_INT, 0, i, MPI_COMM_WORLD, &s_req[i]);
						//t = MPI_Wtime();
						MPI_Waitany(NSused, r_req, &index, status);
						//t_wait += MPI_Wtime()-t;
						hipMemcpyAsync(d_Dist+n*bps*B*index, Dist+n*bps*B*index, n*bps*B*sizeof(int), hipMemcpyHostToDevice, streams[i]); 
					}
					if(!done[i])finish = false;
				}
			}
			for(int i=0; i<NSused-NSused2; i++){
				//t = MPI_Wtime();
				MPI_Waitany(NSused, r_req, &index, status);
				//t_wait += MPI_Wtime()-t;
				hipMemcpyAsync(d_Dist+n*bps*B*index, Dist+n*bps*B*index, n*bps*B*sizeof(int), hipMemcpyHostToDevice, streams[NS-1]); 
			}
			//t = MPI_Wtime();
			for(int i=0; i<NSused2; i++)MPI_Wait(&s_req[i], &status[i]);
			//t_wait += MPI_Wtime()-t;
			for(int i=0; i<NS; i++)hipStreamSynchronize(streams[i]);
		}
		MPI_Barrier(MPI_COMM_WORLD);
	}
	//printf("total time: %f\n", MPI_Wtime()-t_total);
	//printf("wait time: %f\n", t_wait);
	hipDeviceSynchronize();
	if(rank==0){
		//t = MPI_Wtime();
		output(argv[2]);
		//printf("output time: %f\n", MPI_Wtime()-t);
	}
	for(int i=0; i<NS; i++)hipStreamDestroy(streams[i]);
	hipFree(d_Dist);
	hipHostFree(Dist);
	MPI_Finalize();
	return 0;
}
